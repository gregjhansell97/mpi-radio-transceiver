#include "hip/hip_runtime.h"
#include<assert.h>
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>

#include "./cuda_structs.h"

// cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Counts the number of cuda devices
// Returns: the number of cuda devices
int get_cuda_device_count() {
    int device_count;
    hipError_t cuda_status = hipGetDeviceCount(&device_count);
    // check for device count failure
    if(cuda_status != hipSuccess) {
        printf("Unable to determine cuda device count, error is %d, count is %d\n", 
                cuda_status, device_count);
        exit(-1);
    }
    return device_count;
}

// Sets the current cuda device
void set_cuda_device(int rank, int cuda_device) {
    hipError_t cuda_status =  hipSetDevice(cuda_device);
    // check for device count failure
    if( cuda_status != hipSuccess) {
        printf("Unable to have rank %d set to cuda device %d, error is %d \n",
                rank, cuda_device, cuda_status);
        exit(-1);
    }
}

// allocates memory for cuda
void allocate_cuda_memory(char** data, const size_t size) {
    hipMallocManaged(data, size);
}

// wait for all parallel gpu calculations to finish
void synchronize_cuda_devices() {
    hipDeviceSynchronize();
}

// frees allocated cuda memory
void free_cuda_memory(char* data) {
    hipFree(data);
}

__global__ void deliver_mpi_msg_kernel(
        const size_t num_trxs,
        const size_t device_data_size,
        const size_t mail_size,
        const size_t max_buffer_size,
        const size_t packet_size,
        const double latency,
        const double current_time,
        char* raw_mpi_msg, char* raw_device_data) {
    // this is where things get fast!
    MPIMsg* mpi_msg = (MPIMsg*)(raw_mpi_msg);
    size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
    const size_t step = blockDim.x * gridDim.x; // total threads in process
    double mag;
    double dx;
    double dy;
    Mail* head;
    Mail* tail;
    for(; i < num_trxs; i += step) {
        if(i == 1) {
            printf("ITERATING THROUGH\n");
        }

        DeviceData* d = (DeviceData*)(raw_device_data + i*device_data_size);
        //sanity check

        if(d->buffer_size + mpi_msg->size > max_buffer_size) {
            // buffer overflow
            continue;
        }
        if(mpi_msg->sender_rank == d->rank &&
                mpi_msg->sender_id == d->id) {
            // don't send to self
            continue;
        }
        // calculate distance
        mag = mpi_msg->send_range + d->recv_range;
        dx = mpi_msg->send_x - d->x;
        dy = mpi_msg->send_y - d->y;
        if(mag*mag < dx*dx + dy*dy) {
            //  nodes too far away 
            continue;
        }
        // head and tail of queue
        head = (Mail*)((char*)(&d->_mailbox) + (d->_head)*mail_size);
        tail = (Mail*)((char*)(&d->_mailbox) + (d->_tail)*mail_size);
        // not empty and inteference 
        if(d->buffer_size > 0
                && mpi_msg->send_time - head->send_time < latency) {
            // NOTE: ^^ should be one before tail not head
            // grow leading msg pointer to absorb other msg
            head->size += mpi_msg->size;
            // set head pointer to have interference
            head->interference = true;
            d->buffer_size += mpi_msg->size;
        } else {
            printf("got a new message\n");
            tail->send_time = mpi_msg->send_time;
            tail->interference = (d->last_send_time + latency > current_time);
            tail->size = mpi_msg->size;
            // copy over data from mpi message to tail
            memcpy(&tail->data, &mpi_msg->data, mpi_msg->size);
            // adjust tail to next open spot
            d->_tail = (d->_tail + 1)%max_buffer_size;
            d->buffer_size = d->buffer_size + mpi_msg->size;
        }
    }
}

void deliver_mpi_msg(
        const unsigned long blocks_count,
        const ushort threads_per_block,
        const size_t num_trxs,
        const size_t device_data_size,
        const size_t mail_size,
        const size_t max_buffer_size,
        const size_t packet_size,
        const double latency,
        const double current_time,
        char* raw_mpi_msg, char* raw_device_data) {
    deliver_mpi_msg_kernel<<<blocks_count, threads_per_block>>>(
            num_trxs,
            device_data_size,
            mail_size,
            max_buffer_size,
            packet_size,
            latency,
            current_time,
            raw_mpi_msg, raw_device_data);
    synchronize_cuda_devices();
}
